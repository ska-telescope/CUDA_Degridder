#include "hip/hip_runtime.h"

// Copyright 2019 Adam Campbell, Seth Hall, Andrew Ensor
// Copyright 2019 High Performance Computing Research Laboratory, Auckland University of Technology (AUT)

// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:

// 1. Redistributions of source code must retain the above copyright notice,
// this list of conditions and the following disclaimer.

// 2. Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and/or other materials provided with the distribution.

// 3. Neither the name of the copyright holder nor the names of its
// contributors may be used to endorse or promote products derived from this
// software without specific prior written permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.

#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <hip/hip_math_constants.h>
#include <>
#include <numeric>

#include "degridder.h"

void init_config(Config *config)
{
	// Single dimension of grid (dirty residual image)
	config->grid_size = 18000;
	
	config->right_ascension = true;

	config->force_zero_w_term = false;
	
	config->cell_size = 6.39708380288950e-6;
	
	config->frequency_hz = 100e6;
	
	// Single dimension of basic convolution kernel
	config->kernel_size = 9;

	// Specify the number of kernels used in w projection
	config->num_wproj_kernels = 339;

	config->max_w = 7083.386050;

	config->w_scale = pow(config->num_wproj_kernels - 1, 2.0) / config->max_w;
	
	// Kernel oversampling factor
	config->oversampling = 4; // Oxford configuration
	
	// Used to convert visibility uvw coordinates into grid coordinates
	config->uv_scale = config->grid_size * config->cell_size;
	
	// Number of visibilities to process
	config->num_visibilities = 100;
	
	// File location to load grid
	config->grid_real_source_file = "../unit_test_data/ModelImage_18000.csv";

	// File location to load pre-calculated w-projection kernel
	config->kernel_real_source_file = "../unit_test_data/kernels/el82-el70_kernel_real.csv";
	config->kernel_imag_source_file = "../unit_test_data/kernels/el82-el70_kernel_imag.csv";

	// Specify file which holds the supports for all kernels
	config->kernel_support_file = "../unit_test_data/kernels/el82-el70_kernel_supports.csv";

	// File location to load visibility uvw coordinates  
	config->visibility_source_file = "../unit_test_data/el82-70_unit_test.csv";   
	
	// File location to store extracted visibilities  
	config->visibility_dest_file = "../el82-70_output_degridder.csv";

	// Number of CUDA threads per block - this is GPU specific
	config->gpu_max_threads_per_block = 1024;

	//Number of CUDA threads per block dimension in x and y - this is GPU specific and used for FFT and CC
	config->gpu_max_threads_per_block_dimension = 32;

	// Enable/disable CUDA timing of degridding kernel 
	config->time_degridding = true;

	// Perform convolution correction and FFT pre-processing on input grid
	config->conv_correction_and_fft = true;

	config->conv_correction_performed = false;
}
void execute_degridding(Config *config, Complex *grid, 
	Visibility *vis_uvw, Complex *vis_intensities, int num_visibilities, double2* prolate, 
	Complex *kernel, int2 *kernel_supports, int num_kernel_samples)
{
	bool perform_cc_and_fft = config->conv_correction_and_fft;
	bool cc_performed = config->conv_correction_performed;
	hipEvent_t start, stop;
	// Handles for GPU memory
	double2 *d_grid;
	double2 *d_kernel;
	double3 *d_vis_uvw;
	double2 *d_vis;
	double2 *d_prolate;
	int2 *d_supports;


	printf("Binding grid to GPU...\n");	
	// Allocate and copy grid to GPU
	int grid_size_square = config->grid_size * config->grid_size;
	CUDA_CHECK_RETURN(hipMalloc(&d_grid, sizeof(double2) * grid_size_square));
	CUDA_CHECK_RETURN(hipMemcpy(d_grid, grid, sizeof(double2) * grid_size_square, hipMemcpyHostToDevice));
	hipDeviceSynchronize();


	// Perform convolution correction on host based grid
	if(perform_cc_and_fft && !cc_performed)
	{
		CUDA_CHECK_RETURN(hipMalloc(&d_prolate, sizeof(double2) * config->grid_size/2));
		CUDA_CHECK_RETURN(hipMemcpy(d_prolate, prolate, sizeof(double2) * config->grid_size/2, hipMemcpyHostToDevice));
		hipDeviceSynchronize();

		int max_threads_per_block_dimension = min(config->gpu_max_threads_per_block_dimension, config->grid_size);
		int num_blocks_per_dimension = (int) ceil((double) config->grid_size / max_threads_per_block_dimension);
		dim3 cc_blocks(num_blocks_per_dimension, num_blocks_per_dimension, 1);
		dim3 cc_threads(max_threads_per_block_dimension, max_threads_per_block_dimension, 1);
		// Perform 2D FFT shift

		if(config->time_degridding)
		{	
			hipEventCreate(&start);
			hipEventCreate(&stop);
			hipEventRecord(start);
		}

		execute_convolution_correction<<<cc_blocks, cc_threads>>>(d_grid, d_prolate, config->grid_size);

		if(config->time_degridding)
		{
			hipEventRecord(stop);
			hipEventSynchronize(stop);
			float milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start, stop);
			printf(">>> GPU accelerated Convolution Correction completed in %f milliseconds...\n\n", milliseconds);
		}

		hipDeviceSynchronize();
		printf(">>> CONVOLUTION CORRECTION DONE \n...");
		config->conv_correction_performed = true;
		CUDA_CHECK_RETURN(hipFree(d_prolate));
	}


	// Perform FFT on device based grid
	if(perform_cc_and_fft)
	{
		printf(">>> PERFORMING FFT\n...");
		if(config->time_degridding)
		{	
			hipEventCreate(&start);
			hipEventCreate(&stop);
			hipEventRecord(start);
		}
		execute_CUDA_FFT(config, d_grid);
		if(config->time_degridding)
		{
			hipEventRecord(stop);
			hipEventSynchronize(stop);
			float milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start, stop);
			printf(">>> GPU accelerated FFT completed in %f milliseconds...\n\n", milliseconds);
		}
	}

	printf("Grid bound to GPU...\n");
	// Allocate and copy kernel to device
	//int oversampled_half_kernel = config->oversampling * ((config->kernel_size / 2) + 1);
	//int total_kernel_size = oversampled_half_kernel * oversampled_half_kernel;

	CUDA_CHECK_RETURN(hipMalloc(&d_kernel, sizeof(double2) * num_kernel_samples));
	CUDA_CHECK_RETURN(hipMemcpy(d_kernel, kernel, sizeof(double2) * num_kernel_samples,
		hipMemcpyHostToDevice));
	hipDeviceSynchronize();
	//bind the supports array

	CUDA_CHECK_RETURN(hipMalloc(&d_supports, sizeof(int2) * config->num_wproj_kernels));
	CUDA_CHECK_RETURN(hipMemcpy(d_supports, kernel_supports, sizeof(int2) * config->num_wproj_kernels,
		hipMemcpyHostToDevice));
	hipDeviceSynchronize();

	printf("Kernels bound to GPU...\n");
	// Allocate and copy visibility uvw to device
	CUDA_CHECK_RETURN(hipMalloc(&d_vis_uvw, sizeof(double3) * num_visibilities));
	CUDA_CHECK_RETURN(hipMemcpy(d_vis_uvw, vis_uvw, sizeof(double3) * num_visibilities,
		hipMemcpyHostToDevice));
	hipDeviceSynchronize();
	printf("Vis UVW bound to GPU...\n");

	// Allocate memory on device for storing extracted complex visibilities
	CUDA_CHECK_RETURN(hipMalloc(&d_vis, sizeof(double2) * num_visibilities));
	hipDeviceSynchronize();
	printf("Vis memory allocated on GPU...\n");

	int max_threads_per_block = min(config->gpu_max_threads_per_block, num_visibilities);
	int num_blocks = (int) ceil((double) num_visibilities / max_threads_per_block);
	dim3 kernel_blocks(num_blocks, 1, 1);
	dim3 kernel_threads(max_threads_per_block, 1, 1);

	printf(">>> Going to use %d number of blocks, %d threads, for %d visibilities...\n",
		num_blocks, max_threads_per_block, num_visibilities);

	// Optional timing functionality
	if(config->time_degridding)
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start);
	}



	printf("Beginning kernel...\n");
	// Execute degridding kernel
	degridding<<<kernel_blocks, kernel_threads>>>(d_grid, d_kernel, d_supports, 
		d_vis_uvw, d_vis, num_visibilities, config->oversampling,
		config->grid_size, config->uv_scale, config->w_scale);
	hipDeviceSynchronize();
	printf("Finished kernel...\n");

	// Optional report on timing
	if(config->time_degridding)
	{
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		printf(">>> GPU accelerated degridding completed in %f milliseconds...\n\n", milliseconds);
	}

	// Copy back predicted complex visibilities to host
	CUDA_CHECK_RETURN(hipMemcpy(vis_intensities, d_vis,
		num_visibilities * sizeof(double2), hipMemcpyDeviceToHost));
	hipDeviceSynchronize();

	// Clean up
	CUDA_CHECK_RETURN(hipFree(d_grid));
	CUDA_CHECK_RETURN(hipFree(d_kernel));
	CUDA_CHECK_RETURN(hipFree(d_vis_uvw));
	CUDA_CHECK_RETURN(hipFree(d_vis));
	CUDA_CHECK_RETURN(hipFree(d_supports));
	CUDA_CHECK_RETURN(hipDeviceReset());
}

int read_kernel_supports(Config *config, int2 *kernel_supports)
{
	int total_kernel_samples_needed = 0;

	FILE *kernel_support_file = fopen(config->kernel_support_file,"r");

	if(kernel_support_file == NULL)
	{
		return -1;
	}


	for(int plane_num = 0; plane_num < config->num_wproj_kernels; ++plane_num)
	{
		fscanf(kernel_support_file,"%d\n",&(kernel_supports[plane_num].x));
		kernel_supports[plane_num].y = total_kernel_samples_needed;
		total_kernel_samples_needed += (int)pow((kernel_supports[plane_num].x + 1) * config->oversampling, 2.0);

		//printf("%d) %d support with offset at %d \n",plane_num,kernel_supports[plane_num].x,kernel_supports[plane_num].y);
	}

	fclose(kernel_support_file);
	return total_kernel_samples_needed;
}
 

__global__ void fftshift_2D(double2 *grid, const int width)
{
    int row_index = threadIdx.y + blockDim.y * blockIdx.y;
    int col_index = threadIdx.x + blockDim.x * blockIdx.x;
 
    if(row_index >= width || col_index >= width)
        return;
 
    double a = 1 - 2 * ((row_index + col_index) & 1);
    grid[row_index * width + col_index].x *= a;
    grid[row_index * width + col_index].y *= a;
}

__device__ double2 complex_mult(const double2 z1, const double2 z2)
{
	return make_double2(z1.x * z2.x - z1.y * z2.y, z1.y * z2.x + z1.x * z2.y);
}


__global__ void degridding(const double2 *grid, const double2 *kernel, const int2 *supports,
	const double3 *vis_uvw, double2 *vis, const int num_vis, const int oversampling,
	const int grid_size, const double uv_scale, const double w_scale)
{
	const unsigned int vis_index = blockIdx.x * blockDim.x + threadIdx.x;

	if(vis_index >= num_vis)
		return;

	// Represents index of w-projection kernel in supports array
	const int plane_index = (int) round(sqrt(fabs(vis_uvw[vis_index].z * w_scale)));

	// Scale visibility uvw into grid coordinate space
	const double2 grid_coord = make_double2(
		vis_uvw[vis_index].x * uv_scale,
		vis_uvw[vis_index].y * uv_scale
	);

	const int half_grid_size = grid_size / 2;
	const int half_support = supports[plane_index].x;

	double conjugate = (vis_uvw[vis_index].z < 0.0) ? -1.0 : 1.0;
	conjugate *= -1.0;
	//HAD TO REVERSE THE CONJUGATE FOR DEGRIDDING..
	//double conjugate = (vis_uvw[vis_index].z > 0.0) ? -1.0 : 1.0;


	const double2 snapped_grid_coord = make_double2(
		round(grid_coord.x * oversampling) / oversampling,
		round(grid_coord.y * oversampling) / oversampling
	);

	const double2 min_grid_point = make_double2(
		ceil(snapped_grid_coord.x - half_support),
		ceil(snapped_grid_coord.y - half_support)
	);

	const double2 max_grid_point = make_double2(
		floor(snapped_grid_coord.x + half_support),
		floor(snapped_grid_coord.y + half_support)
	);

	double2 grid_point = make_double2(0.0, 0.0);
	double2 deconvolved = make_double2(0.0, 0.0);
	double2 predicted_vis = make_double2(0.0, 0.0);
	double2 kernel_sample = make_double2(0.0, 0.0);
	int2 kernel_uv_index = make_int2(0, 0);

	int grid_index = 0;
	int kernel_index = 0;
	int w_kernel_offset = supports[plane_index].y;

	// Iteratively extract predicted visibility from grid
	for(int grid_v = min_grid_point.y; grid_v <= max_grid_point.y; ++grid_v)
	{	
		kernel_uv_index.y = abs((int)round((grid_v - snapped_grid_coord.y) * oversampling));
		
		for(int grid_u = min_grid_point.x; grid_u <= max_grid_point.x; ++grid_u)
		{
			kernel_uv_index.x = abs((int)round((grid_u - snapped_grid_coord.x) * oversampling));

			kernel_index = w_kernel_offset + kernel_uv_index.y * (half_support + 1)
				* oversampling + kernel_uv_index.x;
			kernel_sample = make_double2(kernel[kernel_index].x, kernel[kernel_index].y  * conjugate);

			grid_index = (grid_v + half_grid_size) * grid_size + (grid_u + half_grid_size);

			double2 gridPoint = make_double2(grid[grid_index].x, grid[grid_index].y);

			deconvolved = complex_mult(gridPoint, kernel_sample);

			predicted_vis.x += deconvolved.x;
			predicted_vis.y += deconvolved.y;
		}
	}


	vis[vis_index] = make_double2(predicted_vis.x, predicted_vis.y);
}
 

void save_visibilities(Config *config, Visibility *vis_uvw, Complex *vis_intensity)
{
	FILE *vis_file = fopen(config->visibility_dest_file, "w");
	
	if(vis_file == NULL)
	{
		printf("Unable to open file...\n");
		return; // unsuccessfully saved visibility data
	}
	
	// Define the number of processed visibilities
	fprintf(vis_file, "%d\n", config->num_visibilities);
	
	double meters_to_wavelengths = config->frequency_hz / C;
	Visibility current_vis;
	Complex current_intensity;
	
	for(int vis_index = 0; vis_index < config->num_visibilities; ++vis_index)
	{
		current_vis = vis_uvw[vis_index];
		current_intensity = vis_intensity[vis_index];
		
		current_vis.u /= meters_to_wavelengths;
		current_vis.v /= meters_to_wavelengths;
		current_vis.w /= meters_to_wavelengths;
		
		if(config->right_ascension)
		{
			current_vis.u *= -1.0;
			current_vis.w *= -1.0;
		}
		
		// u, v, w, vis(real), vis(imag), weighting
		fprintf(vis_file, "%.15f %.15f %.15f %.15f %.15f %.15f\n", 
			current_vis.u,
			current_vis.v,
			current_vis.w,
			current_intensity.real,
			current_intensity.imag,
			1.0); // static weight (for now)
	}
	
	fclose(vis_file);
}

bool load_kernel(Config *config, Complex *kernel, int2 *kernel_supports)
{
	FILE *kernel_real_file = fopen(config->kernel_real_source_file, "r");
	FILE *kernel_imag_file = fopen(config->kernel_imag_source_file, "r");
	
	if(kernel_real_file == NULL || kernel_imag_file == NULL)
	{
		printf("ERROR >>> Unable to open kernel source files...\n");
		if(kernel_real_file != NULL) fclose(kernel_real_file);
		if(kernel_imag_file != NULL) fclose(kernel_imag_file);
		return false; // unsuccessfully loaded data
	}
	
	int kernel_index = 0;

	for(int plane_num = 0; plane_num < config->num_wproj_kernels; ++plane_num)
	{
		int number_samples_in_kernel = (int) pow((kernel_supports[plane_num].x + 1) * config->oversampling, 2.0);

		for(int sample_number = 0; sample_number < number_samples_in_kernel; ++sample_number)
		{	
			float real = 0.0;
			float imag = 0.0;

			fscanf(kernel_real_file, "%f ", &real);
			fscanf(kernel_imag_file, "%f ", &imag);

			kernel[kernel_index] = (Complex) {.real = real, .imag = imag};
			kernel_index++;
		}

	}
	printf("Total number of samples read %d \n",kernel_index);

	fclose(kernel_real_file);
	fclose(kernel_imag_file);
	return true;
}


bool load_grid(Config *config, Complex *grid)
{
	FILE *grid_real_file = fopen(config->grid_real_source_file, "r");
	
	if(grid_real_file == NULL)
	{
		printf("Unable to open grid files...\n");
		if(grid_real_file != NULL) fclose(grid_real_file);
		return false; // unsuccessfully loaded data
	} 
	
	int grid_size = config->grid_size;
	int grid_index = 0;
	double grid_real = 0.0;
	
	double valuesum = 0.0;

	for(int row_index = 0; row_index < grid_size; ++row_index)
	{
		for(int col_index = 0; col_index < grid_size; ++col_index)
		{
			if(col_index < grid_size-1)
			{
				fscanf(grid_real_file, "%lf,", &grid_real);
			}
			else
			{
				fscanf(grid_real_file, "%lf\n", &grid_real);
			}

			valuesum += grid_real;
			grid_index = row_index * grid_size + col_index;
			grid[grid_index] = (Complex) {.real = grid_real, .imag = 0.0};
		}
	}
	printf("UPDATE >>> Successfully read in %f source intensity sum \n",valuesum);
	fclose(grid_real_file);
	return true; 
}

bool load_visibilities(Config *config, Visibility **vis_uvw, Complex **vis_intensities)
{
	// Attempt to open visibility source file
	FILE *vis_file = fopen(config->visibility_source_file, "r");
	if(vis_file == NULL)
	{
		printf("Unable to open visibility file...\n");
		return false; // unsuccessfully loaded data
	}
	
	// Configure number of visibilities from file
	int num_vis = 0;
	fscanf(vis_file, "%d", &num_vis);
	config->num_visibilities = num_vis;

	// Allocate memory for incoming visibilities
	*vis_uvw = (Visibility*) calloc(num_vis, sizeof(Visibility));
	*vis_intensities = (Complex*) calloc(num_vis, sizeof(Complex));
	if(*vis_uvw == NULL || *vis_intensities == NULL)
	{
		printf("Unable to allocate memory...\n");
		fclose(vis_file);
		return false;
	}
	
	// Load visibility uvw coordinates into memory
	double vis_u = 0.0;
	double vis_v = 0.0;
	double vis_w = 0.0;
	double vis_real = 0.0;
	double vis_imag = 0.0;
	double vis_weight = 0.0;
	double meters_to_wavelengths = config->frequency_hz / C;


	for(int vis_index = 0; vis_index < num_vis; ++vis_index)
	{
		// Discard vis(real), vis(imag), and weighting (for now)
		fscanf(vis_file, "%lf %lf %lf %lf %lf %lf\n", &vis_u, &vis_v,
			&vis_w, &vis_real, &vis_imag, &vis_weight);
	
		(*vis_uvw)[vis_index] = (Visibility) {
			.u = vis_u * meters_to_wavelengths,
			.v = vis_v * meters_to_wavelengths,
			.w = (config->force_zero_w_term) ? 0.0 : vis_w * meters_to_wavelengths 
		};

		if(config->right_ascension)
		{
			(*vis_uvw)[vis_index].u *= -1.0;
			(*vis_uvw)[vis_index].w *= -1.0;
		}
	}
	// Clean up
	fclose(vis_file);
	return true;
}

// Calculates a sample on across a prolate spheroidal
// Note: this is the Fred Schwabb approximation technique
double calc_spheroidal_sample(double nu)
{
    static double p[] = {0.08203343, -0.3644705, 0.627866, -0.5335581, 0.2312756,
        0.004028559, -0.03697768, 0.1021332, -0.1201436, 0.06412774};
    static double q[] = {1.0, 0.8212018, 0.2078043,
        1.0, 0.9599102, 0.2918724};

    int part = 0;
    int sp = 0;
    int sq = 0;
    double nuend = 0.0;
    double delta = 0.0;
    double top = 0.0;
    double bottom = 0.0;

    if(nu >= 0.0 && nu < 0.75)
    {
        part = 0;
        nuend = 0.75;
    }
    else if(nu >= 0.75 && nu < 1.0)
    {
        part = 1;
        nuend = 1.0;
    }
    else
        return 0.0;

    delta = nu * nu - nuend * nuend;
    sp = part * 5;
    sq = part * 3;
    top = p[sp];
    bottom = q[sq];

    for(int i = 1; i < 5; i++)
        top += p[sp+i] * pow(delta, i);
    for(int i = 1; i < 3; i++)
        bottom += q[sq+i] * pow(delta, i);
    return (bottom == 0.0) ? 0.0 : top/bottom;
}

void create_1D_half_prolate(double2 *prolate, int grid_size, double cell_size)
{
	int grid_half_size = grid_size / 2;
	double nu = 0.0;
	double taper = 0.0;
	double l_square = 0.0;

	for(int index = 0; index < grid_half_size; ++index)
	{
		nu = ((double)index / (double)grid_half_size);
		taper = calc_spheroidal_sample(nu);
		prolate[index].x = taper;
		l_square = pow((double)index * cell_size, 2.0);
		prolate[index].y = l_square;
	}
}


__global__ void execute_convolution_correction(double2 *grid, const double2 *prolate, const int grid_size)
{
	const int row_index = threadIdx.y + blockDim.y * blockIdx.y;
    const int col_index = threadIdx.x + blockDim.x * blockIdx.x;

    if(row_index >= grid_size || col_index >= grid_size)
    	return;

    const int grid_index = row_index * grid_size + col_index;
    const int half_grid_size = grid_size / 2;

    const double taper = prolate[abs(col_index - half_grid_size)].x * prolate[abs(row_index - half_grid_size)].x;

    const double l = prolate[abs(col_index - half_grid_size)].y;
    const double m = prolate[abs(row_index - half_grid_size)].y;

    grid[grid_index].x = (fabs(taper) > (1E-10)) ? grid[grid_index].x / (taper * sqrt(1.0 - l - m)) : 0.0;
}

void execute_CUDA_FFT(Config *config, double2 *grid)
{
	int grid_size = config->grid_size;

	int max_threads_per_block_dimension = min(config->gpu_max_threads_per_block_dimension, grid_size);
	int num_blocks_per_dimension = (int) ceil((double) grid_size / max_threads_per_block_dimension);
	dim3 shift_blocks(num_blocks_per_dimension, num_blocks_per_dimension, 1);
	dim3 shift_threads(max_threads_per_block_dimension, max_threads_per_block_dimension, 1);

	printf("Shifting grid data for 2D FFT...\n");
	// Perform 2D FFT shift
	fftshift_2D<<<shift_blocks, shift_threads>>>(grid, grid_size);
	hipDeviceSynchronize();

	printf("Performing 2D FFT...\n");
	// Perform 2D FFT
	hipfftHandle fft_plan;
	CUFFT_SAFE_CALL(hipfftPlan2d(&fft_plan, grid_size, grid_size, HIPFFT_Z2Z));
	CUFFT_SAFE_CALL(hipfftExecZ2Z(fft_plan, grid, grid, HIPFFT_FORWARD));
	hipDeviceSynchronize();

	printf("Shifting grid data back into place...\n");
	// Perform 2D FFT shift back
	fftshift_2D<<<shift_blocks, shift_threads>>>(grid, grid_size);
	hipDeviceSynchronize();
}

void clean_up(Complex **grid, Visibility **vis_uvw, Complex **vis_intensities, Complex **kernel, int2 **kernel_supports, double2 **prolate)
{
	printf(">>> Cleaning up allocated host memory...\n");
	if(*grid) 			 free(*grid);
	if(*vis_uvw) 	 	 free(*vis_uvw);
	if(*vis_intensities) free(*vis_intensities);
	if(*kernel) 		 free(*kernel);
	if(*kernel_supports) free(*kernel_supports);
	if(*prolate) 		 free(*prolate);
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void check_cuda_error_aux(const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;

	printf(">>> CUDA ERROR: %s returned %s at %s : %u ",statement, file, hipGetErrorString(err), line);
	exit(EXIT_FAILURE);
}

static void cufft_safe_call(hipfftResult err, const char *file, const int line)
{
    if( HIPFFT_SUCCESS != err) {
		printf("CUFFT error in file '%s', line %d\nerror %d: %s\nterminating!\n",
			__FILE__, __LINE__, err, cuda_get_error_enum(err));
		hipDeviceReset();
    }
}

static const char* cuda_get_error_enum(hipfftResult error)
{
    switch (error)
    {
        case HIPFFT_SUCCESS:
            return "HIPFFT_SUCCESS";

        case HIPFFT_INVALID_PLAN:
            return "HIPFFT_INVALID_PLAN";

        case HIPFFT_ALLOC_FAILED:
            return "HIPFFT_ALLOC_FAILED";

        case HIPFFT_INVALID_TYPE:
            return "HIPFFT_INVALID_TYPE";

        case HIPFFT_INVALID_VALUE:
            return "HIPFFT_INVALID_VALUE";

        case HIPFFT_INTERNAL_ERROR:
            return "HIPFFT_INTERNAL_ERROR";

        case HIPFFT_EXEC_FAILED:
            return "HIPFFT_EXEC_FAILED";

        case HIPFFT_SETUP_FAILED:
            return "HIPFFT_SETUP_FAILED";

        case HIPFFT_INVALID_SIZE:
            return "HIPFFT_INVALID_SIZE";

        case HIPFFT_UNALIGNED_DATA:
            return "HIPFFT_UNALIGNED_DATA";
    }

    return "<unknown>";
}


/***************************************
*      UNIT TESTING FUNCTIONALITY      *
***************************************/

//Unit test for l
double unit_test_gpu_convolution_correction(Complex *grid, int grid_size, double cell_size)
{
	double2 *d_grid;
	double2 *d_prolate;

	double2 *prolate = (double2*) calloc(grid_size / 2,sizeof(double2));
	create_1D_half_prolate(prolate, grid_size, cell_size);

	int grid_size_square = grid_size * grid_size;
	Complex *outputGrid = (Complex*) calloc(grid_size_square, sizeof(Complex));


	printf("UNIT TEST >>> COPYING PROLATE TO GPU BUFFER..\n");
	CUDA_CHECK_RETURN(hipMalloc(&d_prolate, sizeof(double2) * grid_size/2));
	CUDA_CHECK_RETURN(hipMemcpy(d_prolate, prolate, sizeof(double2) * grid_size/2, hipMemcpyHostToDevice));
	hipDeviceSynchronize();

	printf("UNIT TEST >>> COPYING GRID TO GPU...\n");
	
	CUDA_CHECK_RETURN(hipMalloc(&d_grid, sizeof(Complex) * grid_size_square));
	CUDA_CHECK_RETURN(hipMemcpy(d_grid, grid, sizeof(Complex) * grid_size_square, hipMemcpyHostToDevice));
	hipDeviceSynchronize();

	printf("UNIT TEST >>> EXECUTING CPU CONVOLUTION CORRECTION...\n");
	execute_convolution_correction_cpu(grid, grid_size, cell_size);

	printf("UNIT TEST >>> EXECUTING GPU CONVOLUTION CORRECTION..\n");

	int max_threads_per_block_dimension = min(32, grid_size);
	int num_blocks_per_dimension = (int) ceil((double) grid_size / max_threads_per_block_dimension);
	dim3 cc_blocks(num_blocks_per_dimension, num_blocks_per_dimension, 1);
	dim3 cc_threads(max_threads_per_block_dimension, max_threads_per_block_dimension, 1);
	execute_convolution_correction<<<cc_blocks, cc_threads>>>(d_grid, d_prolate, grid_size);
	hipDeviceSynchronize();
	printf("UNIT TEST >>> CONVOLUTION CORRECTION DONE, COPYING GRID BACK ...\n");

	CUDA_CHECK_RETURN(hipMemcpy(outputGrid, d_grid,
		grid_size_square * sizeof(Complex), hipMemcpyDeviceToHost));
	hipDeviceSynchronize();

	printf("UNIT TEST >>> CALCULATE RMSE OF GPU vs GPU ...\n");

	double sumSquareDiff;
	double sumSquareCPU;
	for(int i = 0; i < grid_size_square; i++)
	{	sumSquareDiff += pow(abs(grid[i].real-outputGrid[i].real)+abs(grid[i].imag-outputGrid[i].imag),2.0);
		sumSquareCPU += pow(abs(grid[i].real)+abs(grid[i].imag),2.0);
	}

	double rmse = sqrt(sumSquareDiff) / sqrt(sumSquareCPU);

	grid = &(*outputGrid);

	printf("UNIT TEST >>> CLEAN UP USED MEMORY ...\n");
	CUDA_CHECK_RETURN(hipFree(d_prolate));
	CUDA_CHECK_RETURN(hipFree(d_grid));
	free(outputGrid);

	return rmse;
}


// TODO: This could be parallelized...
void execute_convolution_correction_cpu(Complex *grid, double grid_size, double cell_size)
{
	double grid_half_size = grid_size / 2;
	double nu_y = 0.0;
	double nu_x = 0.0;
	double taper_y = 0.0;
	double taper = 0.0;
	double l = 0.0;
	double m = 0.0;
	int grid_index = 0;
	for(int row_index = 0; row_index < grid_size; ++row_index)
	{
		nu_y = fabs((row_index - grid_half_size) / grid_half_size);
		taper_y = calc_spheroidal_sample(nu_y);
		m = pow((row_index - grid_half_size) * cell_size, 2.0);

		for(int col_index = 0; col_index < grid_size; ++col_index)
		{
			nu_x = fabs((col_index - grid_half_size) / grid_half_size);
			taper = taper_y * calc_spheroidal_sample(nu_x);
			grid_index = row_index * grid_size + col_index;

			
			if(fabs(taper) > (1E-10))
			{
				l = pow((col_index - grid_half_size) * cell_size, 2.0);
				grid[grid_index].real /= (taper * sqrt(1.0 - l - m));
			}
			else
				grid[grid_index].real = 0.0;
		}
	}
}

void unit_test_init_config(Config *config)
{
	config->grid_size 							= 18000;
	config->right_ascension 					= true;
	config->cell_size 							= 6.39708380288950e-6;
	config->frequency_hz 						= 100e6;
	config->oversampling 						= 4;
	config->uv_scale 							= config->grid_size * config->cell_size;
	config->grid_real_source_file 				= "../unit_test_data/ModelImage_18000.csv";
	config->kernel_support_file         		= "../unit_test_data/kernels/el82-el70_kernel_supports.csv";
	config->kernel_real_source_file 			= "../unit_test_data/kernels/el82-el70_kernel_real.csv";
	config->kernel_imag_source_file 			= "../unit_test_data/kernels/el82-el70_kernel_imag.csv";
	config->visibility_source_file 				= "../unit_test_data/el82-70_unit_test.csv";
	config->visibility_dest_file 				= "../el82-70_unit_test_output.csv";
	config->gpu_max_threads_per_block 			= 1024;
	config->gpu_max_threads_per_block_dimension = 32;
	config->time_degridding 					= true;
	config->conv_correction_and_fft 			= true;
	config->conv_correction_performed 			= false;
	config->force_zero_w_term 					= false;
	config->num_wproj_kernels 					= 339;
	config->max_w 								= 7083.386050;
	config->w_scale 							= pow(config->num_wproj_kernels - 1, 2.0) / config->max_w;
}

double unit_test_output_visibilities(Config *config, Visibility *vis_uvw, Complex *vis_intensities)
{
	// Attempt to open visibility source file
	FILE *vis_file = fopen(config->visibility_source_file, "r");
	if(vis_file == NULL)
	{
		printf("Unable to open visibility file...\n");
		return DBL_MAX; // unsuccessfully loaded data
	}
	
	// Configure number of visibilities from file
	int num_vis = 0;
	fscanf(vis_file, "%d", &num_vis);
	config->num_visibilities = num_vis;

	// Load visibility uvw coordinates into memory
	double vis_u = 0.0;
	double vis_v = 0.0;
	double vis_w = 0.0;
	double vis_real = 0.0;
	double vis_imag = 0.0;
	double vis_weight = 0.0;

	double max_error = 0.0;
	for(int vis_index = 0; vis_index < num_vis; ++vis_index)
	{
		// Discard vis(real), vis(imag), and weighting (for now)
		fscanf(vis_file, "%lf %lf %lf %lf %lf %lf\n", &vis_u, &vis_v,
			&vis_w, &vis_real, &vis_imag, &vis_weight);
		
		double current_diff = sqrt(pow(vis_real - vis_intensities[vis_index].real, 2.0)
	  		+ pow(vis_imag - vis_intensities[vis_index].imag, 2.0));

		if(current_diff > max_error)
			max_error = current_diff;
	}
	// Clean up
	fclose(vis_file);
	return max_error;
}
